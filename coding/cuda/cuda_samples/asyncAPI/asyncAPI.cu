#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value;
}

bool correct_output(int *data, const int n, const int x)
{
    for (int i = 0; i < n; i++)
        if (data[i] != x)
        {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }

    return true;
}

int main()
{
    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    hipHostMalloc((void **)&a, nbytes, hipHostMallocDefault);
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    hipMalloc((void**)&d_a, nbytes);
    hipMemset(d_a, 0, nbytes);

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n/threads.x, 1);

    // issue work to the GPU
    hipMemcpy(d_a,a,nbytes,hipMemcpyHostToDevice);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);

    // check the output for correctness
    bool bFinalResults = correct_output(a, n, value);

    // release resources
    hipHostFree(a);
    hipFree(d_a);

    exit(0);
}